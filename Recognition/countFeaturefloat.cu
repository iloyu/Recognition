#include "hip/hip_runtime.h"
#include "countFeature.cuh"
#define L2HYS_EPSILON 		0.01f
#define L2HYS_EPSILONHYS	1.0f
#define L2HYS_CLIP			0.2f
#define data_h2y            30
__global__ void countCell(uchar *in,float *out,int *d_ANG,int *d_Mag,float *c_ANG,float *c_Mag,float *p_ANG,float *p_Mag,int ImageHeight,int Imagewidth)
{

    int xx=blockIdx.x*blockDim.x+threadIdx.x;
    int yy=blockIdx.y*blockDim.y+threadIdx.y;
    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    long id=xx+yy*Imagewidth;
    int idx=tidy*Windowx+tidx;
     __shared__  float histo[1260 ];//һ��Բ��18������(max(0~17))*����Ŀ��70(ÿ������7��cellÿ��cell 10��bin)+������ţ�max(0~6)��*bin����10��+�����ĸ�bin(max(0~9))=17*70+6*10+9=1259
    if(c_Mag[tidx+tidy*Windowx]>64)
        return;
    memset(histo,0,1260*sizeof(float));//ÿ��������һ��cell������ֱ��ͼ��ʱ����Ҫ��ֱ��ͼ����
    __syncthreads();
    //for(int stridex=0,stridey=0;stridex<gridDim.x, )
        if(xx<Imagewidth&&yy<ImageHeight)
        {
            float t_fm_nbin=p_ANG[yy*ImageHeight+xx]-c_ANG[tidy*Windowy+tidx];
            while(t_fm_nbin<0)
            t_fm_nbin+=Pi;
            int t_nm_nbin=(int)(t_fm_nbin*10/Pi);
            if(tidx<128&&tidy<128)
                atomicAdd(& (histo[d_ANG[tidy*Windowy+tidx]*70+d_Mag[tidy*Windowy+tidx]*10+t_nm_nbin]),p_Mag[yy*ImageHeight+xx]);
            __syncthreads();
            out[d_ANG[tidy*Windowy+tidx]*70+d_Mag[tidy*Windowy+tidx]*10+t_nm_nbin+(blockIdx.x+blockIdx.y*gridDim.x)*1260]=histo[d_ANG[tidy*Windowy+tidx]*70+d_Mag[tidy*Windowy+tidx]*10+t_nm_nbin];

            }
        
    
        }

__global__ void smoothcell(float *in,float *out){
    int t_nleft,t_nright;
    t_nleft=(threadIdx.x-1+10)%10;
    t_nright=(threadIdx.x+1)%10;
    float *t_ptemp,t_ftemp;
    t_ptemp=in+blockIdx.x*70+blockIdx.y*10;//+threadIdx.y)*0.8f+0.1f*(in+blockIdx.x*70+threadIdx.x*10+t_left)
    t_ftemp=t_ptemp[threadIdx.x]*0.8f+0.1f*t_ptemp[t_nleft]+0.1f*t_ptemp[t_nright];
    __syncthreads();
    out[blockIdx.x*70+blockIdx.y*10+threadIdx.x]=t_ptemp[threadIdx.x];
    __syncthreads();
}

__global__ void countblock(float *in ,float *out)
{
    if(in+70*blockIdx.x+(blockIdx.y+blockIdx.x)*10!=NULL)
   { float *ptr_in=in+70*blockIdx.x+(blockIdx.y+blockIdx.x)*10;//threadIdx.x;//70=һ���Ƕȷ���7��cell��ÿ��cell 10��bin,
    float *ptr_out=out+120*blockIdx.x+30*blockIdx.y+10*blockDim.x;//threadIdx.x;//һ���Ƕȷ���4��block��һ��block3��cell��һ��cell 10��bin,
    //һ��block3��cell��һ��cell 10��bin, 
    ptr_out[threadIdx.x]=ptr_in[threadIdx.x];
	}
    }
    
    


 
__global__ void normalizeL2Hys(float *in,float *out)
{
    int bid=blockIdx.x;
    int tid=threadIdx.x;
    // Sum the vector
    float sum = 0;
    
    float *t_ftemp=in+bid*30;
    float *t_foutemp=out+bid*30;
    sum+=t_ftemp[tid]*t_ftemp[tid];
    __syncthreads();
    // Compute the normalization term
    float norm = 1.0f/(rsqrt(sum) + L2HYS_EPSILONHYS * 30);
    t_foutemp[tid]=t_ftemp[tid]*norm;
    __syncthreads();


}
 extern "C" void countFeaturesfloat(uchar *in,float *out,int *d_ANG,int *d_Mag,float *c_ANG,float *c_Mag,float *p_ANG,float *p_Mag,int Imagewidth,int ImageHeight)
{
    int *device_d_ANG,*device_d_Mag;
    float *device_c_ANG, *device_c_Mag,*device_p_Mag,*device_p_ANG,*device_out,*device_smooth_out,*device_block_out,*device_out_norm;
    uchar *device_in;
    long size_d_window=sizeof(int)*Windowx*Windowy;
    long size_c_window=sizeof(float)*Windowx*Windowy;
    long size_c_pixel=sizeof(float)*ImageHeight*Imagewidth;
    long size_uc_pixel=sizeof(uchar)*ImageHeight*Imagewidth;
    long size_c_cell=sizeof(float)*1260*(ImageHeight/Windowy)*(Imagewidth/Windowx);
    long size_s_cell=sizeof(float)*1260;
    long size_c_block=sizeof(float)*2160;

    checkCudaErrors(hipMalloc((void **)&device_c_ANG,size_c_window));
    checkCudaErrors(hipMalloc((void **)&device_c_Mag,size_c_window));
    checkCudaErrors(hipMalloc((void **)&device_d_ANG,size_d_window));
    checkCudaErrors(hipMalloc((void **)&device_d_Mag,size_d_window));
    checkCudaErrors(hipMalloc((void **)&device_p_ANG,size_c_pixel));
    checkCudaErrors(hipMalloc((void **)&device_p_Mag,size_c_pixel));
    checkCudaErrors(hipMalloc((void **)&device_in,size_uc_pixel));
    checkCudaErrors(hipMalloc((void **)&device_out,size_c_cell));
    checkCudaErrors(hipMalloc((void **)&device_smooth_out,size_s_cell));
    checkCudaErrors(hipMalloc((void **)&device_block_out,size_c_block));
    checkCudaErrors(hipMalloc((void **)&device_out_norm,size_c_block));

    checkCudaErrors(hipMemcpy(device_c_ANG,c_ANG,size_c_window,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_c_Mag,c_Mag,size_c_window,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_d_Mag,d_Mag,size_d_window,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_d_ANG,d_ANG,size_d_window,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_p_Mag,p_Mag,size_c_pixel,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_p_ANG,p_ANG,size_c_pixel,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_in,in,size_uc_pixel,hipMemcpyHostToDevice));

    checkCudaErrors(hipMemset(device_out,0,size_c_cell));
    checkCudaErrors(hipMemset(device_smooth_out,0,size_s_cell));
    checkCudaErrors(hipMemset(device_block_out,0,size_c_block));
    checkCudaErrors(hipMemset(device_out_norm,0,size_c_block));

    long shared=sizeof(int)*1260;
    long h_windowx=iDivUp(Imagewidth,Windowx);
    long h_windowy=iDivUp( ImageHeight,Windowy);
    dim3 blocks(h_windowx,h_windowy);//h_windowx=ImageWidth/Windowx,h_windowy=ImageHeight/Windowy
    dim3 threads(Windowx,Windowy);//ÿһ���߳̿����һ��cell��������
    countCell<<<blocks,threads>>>(device_in, device_out, device_d_ANG,device_d_Mag,device_c_ANG, device_c_Mag, device_p_ANG, device_p_Mag, Imagewidth,ImageHeight);
    
    dim3 block_smooth(18,7);//һ��cell��18���Ƕȷ���,һ������7��cell��
    dim3 threads_smooth(10);//ÿ��cell 10 ��bin

    dim3 block_b(18,4);//18=m_nANGһ�����ڷ�18������4=һ���Ƕȷ���4��block
    dim3 thread_b(3,10);//3=һ��block����3��cell,10=һ��cell10��bin

    dim3 block_norm(72);//blob������ 18*4=72
    dim3 thread_norm(30);//block�����������ȣ�m_nBIN��
    
    for(int i=0;i<h_windowx;i++)
        for(int j=0;j<h_windowy;j++)
        {       smoothcell<<<block_smooth,threads_smooth>>>(device_out+(i+h_windowx*j)*1260,device_smooth_out);
                countblock<<<block_b,thread_b>>>(device_smooth_out,device_block_out);
                normalizeL2Hys<<<block_norm,thread_norm>>>(device_block_out,device_out_norm);

				checkCudaErrors(hipMemcpy(out+(i+h_windowx*j)*2160*sizeof(float),device_out_norm,size_c_block,hipMemcpyDeviceToHost));
                hipDeviceSynchronize();
    }

    hipFree(device_c_ANG);
    hipFree(device_c_Mag);
    hipFree(device_d_ANG);
    hipFree(device_d_Mag);
    hipFree(device_p_ANG);
    hipFree(device_p_Mag);
    hipFree(device_in);
    hipFree(device_out);
    hipFree(device_smooth_out);
    hipFree(device_block_out);
    hipFree(device_out_norm);



    hipDeviceReset();
    
    
}