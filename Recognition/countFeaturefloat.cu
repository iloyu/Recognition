#include "hip/hip_runtime.h"
#include "countFeature.cuh"
#define stride 12





__global__ void countCell(float *out,float *device_c_ANG,float *device_c_Mag,float *device_p_ANG,float *device_p_Mag,int ImageHeight,int Imagewidth,int *d_mask,int *d_histo_mask,int offset_X,int offset_Y)
{
	int xx=blockIdx.x*blockDim.x+threadIdx.x;
	int yy=blockIdx.y*blockDim.y+threadIdx.y;
	
	int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    
	int off_X=xx+offset_X;
	int off_Y=yy+offset_Y;
	
	__shared__  float histo[1280];//һ��Բ��18������(max(0~17))*����Ŀ��70(ÿ������7��cellÿ��cell 10��bin)+������ţ�max(0~6)��*bin����10��+�����ĸ�bin(max(0~9))=17*70+6*10+9=1259
	
	__shared__  float t_fm_nbin[Windowy][Windowx];
	//__shared__  float temp[Windowy][Windowx];
	__shared__  int  t_nm_nbin[Windowy][Windowx];
	
	      __syncthreads();
		  t_fm_nbin[tidy][tidx]=device_p_ANG[off_Y*Imagewidth+off_X]-device_c_ANG[(yy)*m_nImage+xx];
	    if( t_fm_nbin[tidy][tidx]<0)
         t_fm_nbin[tidy][tidx]+=Pi; 
		  
		 if( t_fm_nbin[tidy][tidx]<0)
         t_fm_nbin[tidy][tidx]+=Pi; 
		  
		 t_nm_nbin[tidy][tidx]=(int)(t_fm_nbin[tidy][tidx]*10/Pi);
	
		 //out[tidy*32+tidx]=t_nm_nbin[tidy][tidx];
		atomicAdd(& (histo[d_histo_mask[yy*m_nImage+xx]+t_nm_nbin[tidy][tidx]]),
			device_p_Mag[(off_Y)*Imagewidth+ off_X]*d_mask[xx+(yy)*m_nImage]); 

		__syncthreads();
		
		atomicAdd(&out[tidy*32+tidx],histo[tidy*32+tidx]);
		
		if(tidy%4==0)
				atomicAdd(&out[1024+(tidy/4)*32+tidx],histo[1024+(tidy/4)*32+tidx]);
		
}

__global__ void smoothcell(float *in,float *out){
    int t_nleft,t_nright;
    t_nleft=(threadIdx.x-1+10)%10;
    t_nright=(threadIdx.x+1)%10;
    float *t_ptemp,t_ftemp[10];
    t_ptemp=in+blockIdx.x*70+blockIdx.y*10;//+threadIdx.y)*0.8f+0.1f*(in+blockIdx.x*70+threadIdx.x*10+t_left)
	/*__syncthreads();*/
	if(t_ptemp)
	t_ftemp[threadIdx.x]=t_ptemp[threadIdx.x]*0.8f+0.1f*t_ptemp[t_nleft]+0.1f*t_ptemp[t_nright];
    __syncthreads();
	out[blockIdx.x*70+blockIdx.y*10+threadIdx.x]=t_ftemp[threadIdx.x];
    __syncthreads();
}

__global__ void countblock(float *in ,float *out)
{
    //if(in+70*blockIdx.x+(blockIdx.y+threadIdx.x)*10!=NULL)
   //{ 
	float *ptr_in=in+70*blockIdx.x+(blockIdx.y+threadIdx.x)*10;//threadIdx.x;//70=һ���Ƕȷ���7��cell��ÿ��cell 10��bin,
    float *ptr_out=out+120*blockIdx.x+30*blockIdx.y+10*threadIdx.x;//threadIdx.x;//һ���Ƕȷ���4��block��һ��block3��cell��һ��cell 10��bin,
    //һ��block3��cell��һ��cell 10��bin, 
    ptr_out[threadIdx.y]=ptr_in[threadIdx.y];
	////}
    }

__global__ void normalizeL2Hys(float *in,float *out)
{
    int bid=blockIdx.x;
    int tid=threadIdx.x;
    // Sum the vector
	__shared__ float sum[72][15];//15*72
   //memset(sum[15],0,15*sizeof(float));
   __syncthreads();
    float *t_ftemp=in+bid*30;
    float *t_foutemp=out+bid*30;
    if(tid<15) sum[bid][tid]=t_ftemp[tid+15]*t_ftemp[tid+15]+t_ftemp[tid]*t_ftemp[tid];

    __syncthreads();

	if(tid<7) sum[bid][tid]+=sum[bid][tid+7];
	 __syncthreads();

	 if(tid<3) sum[bid][tid]+=sum[bid][tid+3];
	 __syncthreads();
	/* if(tid<2) sum[bid][tid]+=sum[bid][tid+2];
	 __syncthreads();*/
	 if(tid==0) sum[bid][tid]=sum[bid][tid]+sum[bid][tid+1]+sum[bid][14]+sum[bid][6]+sum[bid][2];
	 __syncthreads();
    // Compute the normalization term
	
	 float norm = (rsqrt(sum[bid][0]));
	/*if(sum[1]-0<0.000001) norm=0;*/
	 //printf(" %f ",sum[bid][0]);
	//printf(" %f,%f ",sum[7],norm);
	t_foutemp[tid]=t_ftemp[tid]*norm;
    __syncthreads();


}


 extern "C" void countFeaturesfloat(float *out,float *c_ANG,float *c_Mag,float *p_ANG,float *p_Mag,int Imagewidth,int ImageHeight,int *mask,int *histo_mask,int off_x,int off_y)
{
	 float *device_c_ANG, *device_c_Mag,*device_p_Mag,*device_p_ANG;
     int *d_mask,*d_histo_mask;
    //int *device_d_ANG,*device_d_Mag,*d_mask;
    //float *device_c_ANG, *device_c_Mag,*device_p_Mag,*device_p_ANG,*device_out,*device_smooth_out,*device_block_out,*device_out_norm,*device_smooth_in;
    float *device_out,*device_smooth_out,*device_block_out,*device_out_norm,*device_smooth_in;
	//uchar *device_in;
	//void * m_pClassifier;//������ָ��
	//float t_nRes;//SVM�����ĸ���
	//CvMat *t_FeatureMat;
	//CvSVM * t_pSVM = new CvSVM;
	//	t_pSVM->load( "C:\\Users\\Cyj\\Desktop\\123.xml" );
	//	/*m_pClassifier = (void *)t_pSVM;*/
	//t_FeatureMat = cvCreateMat(  1, 2160,CV_32FC1 );

      long size_d_window=sizeof(int)*m_nImage*m_nImage;
    long size_c_window=sizeof(float)*m_nImage*m_nImage;
    long size_c_pixel=sizeof(float)*ImageHeight*Imagewidth;
    
     long size_s_cell=sizeof(float)*1280;
    long size_c_block=sizeof(float)*2160;

    checkCudaErrors(hipMalloc((void **)&device_c_ANG,size_c_window));
    checkCudaErrors(hipMalloc((void **)&device_c_Mag,size_c_window));

    checkCudaErrors(hipMalloc((void **)&device_p_ANG,size_c_pixel));
    checkCudaErrors(hipMalloc((void **)&device_p_Mag,size_c_pixel));
	checkCudaErrors(hipMalloc((void **)&d_mask,size_d_window));
	checkCudaErrors(hipMalloc((void **)&d_histo_mask,size_d_window));
   
    checkCudaErrors(hipMalloc((void **)&device_out,size_s_cell));
    checkCudaErrors(hipMalloc((void **)&device_smooth_out,size_s_cell));
    checkCudaErrors(hipMalloc((void **)&device_block_out,size_c_block));
    checkCudaErrors(hipMalloc((void **)&device_out_norm,size_c_block));
	/* checkCudaErrors(hipMalloc((void **)&device_in,size_uc_pixel));*/

    checkCudaErrors(hipMemcpy(device_c_ANG,c_ANG,size_c_window,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_c_Mag,c_Mag,size_c_window,hipMemcpyHostToDevice));
 
    checkCudaErrors(hipMemcpy(device_p_Mag,p_Mag,size_c_pixel,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_p_ANG,p_ANG,size_c_pixel,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mask,mask,size_d_window,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_histo_mask,histo_mask,size_d_window,hipMemcpyHostToDevice));
	
    checkCudaErrors(hipMemset(device_out,0,size_s_cell));
    checkCudaErrors(hipMemset(device_smooth_out,0,size_s_cell));
    checkCudaErrors(hipMemset(device_block_out,0,size_c_block));
    checkCudaErrors(hipMemset(device_out_norm,0,size_c_block));

   
	int h_windowx=4;
	int h_windowy=4;
	dim3 blocks(4,4);
	dim3 threads(Windowx,Windowy);//ÿһ���߳̿����һ��cell��������
	//countCell<<<blocks,threads>>>(device_in, device_out, device_d_ANG,device_d_Mag,device_c_ANG, device_c_Mag, device_p_ANG, device_p_Mag, ImageHeight,Imagewidth,d_mask);

	
	dim3 block_right(32);
	dim3 thread_right(4,128);

    dim3 block_smooth(18,7);//һ��cell��18���Ƕȷ���,һ������7��cell��
    dim3 threads_smooth(10);//ÿ��cell 10 ��bin

	
    dim3 block_b(18,4);//18=m_nANGһ�����ڷ�18������4=һ���Ƕȷ���4��block
    dim3 thread_b(3,10);//3=һ��block����3��cell,10=һ��cell10��bin

    dim3 block_norm(72);//blob������ 18*4=72
    dim3 thread_norm(30);//block�����������ȣ�m_nBIN��
    


	countCell<<<blocks,threads>>>( device_out, device_c_ANG, device_c_Mag, device_p_ANG, device_p_Mag, ImageHeight,Imagewidth,d_mask,d_histo_mask,off_x,off_y);
	
	smoothcell<<<block_smooth,threads_smooth>>>(device_out,device_smooth_out);
	countblock<<<block_b,thread_b>>>(device_smooth_out,device_block_out);
	normalizeL2Hys<<<block_norm,thread_norm>>>(device_block_out,device_out_norm);

	checkCudaErrors(hipMemcpy(out,device_out_norm,size_c_block,hipMemcpyDeviceToHost));
		 
			
   /* hipFreeArray(cuArray_Mag);
	   hipFreeArray(cuArray_ANG);*/
    hipFree(device_c_ANG);
    hipFree(device_c_Mag);
    /*hipFree(device_d_ANG);
    hipFree(device_d_Mag);*/
    hipFree(device_p_ANG);
    hipFree(device_p_Mag);
   
    hipFree(device_out);
    hipFree(device_smooth_out);
    hipFree(device_block_out);
    hipFree(device_out_norm);

	hipFree(d_mask);
	hipFree(d_histo_mask);

    hipDeviceReset();
    
    
}